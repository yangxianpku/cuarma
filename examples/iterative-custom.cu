#include "hip/hip_runtime.h"
/* =========================================================================
      Copyright (c) 2015-2017, COE of Peking University, Shaoqiang Tang.

                         -----------------
            cuarma - COE of Peking University, Shaoqiang Tang.
                         -----------------

                  Author Email    yangxianpku@pku.edu.cn

         Code Repo   https://github.com/yangxianpku/cuarma

                      License:    MIT (X11) License
============================================================================= */

/**  @file   iterative-custom.cu
 *   @coding UTF-8
 *   @brief  This tutorial explains the use of iterative solvers in cuarma with custom monitors and initial guesses.
 *   @brief  测试：迭代法求解
 */
 
#include <iostream>

#include "head_define.h"

#include "cuarma/scalar.hpp"
#include "cuarma/vector.hpp"
#include "cuarma/compressed_matrix.hpp"
#include "cuarma/blas/prod.hpp"
#include "cuarma/blas/jacobi_precond.hpp"
#include "cuarma/blas/cg.hpp"
#include "cuarma/blas/bicgstab.hpp"
#include "cuarma/blas/gmres.hpp"
#include "cuarma/io/matrix_market.hpp"

/**
 *  <h1>Defining Custom Monitors Functions for Iterative Solvers</h1>
 *  Custom monitors for the iterative solvers require two ingredients:
 *  First, a structure holding all the auxiliary data we want to reuse in the monitor.
 *  Second, a callback function called by the solver in each iteration.
 *
 *  In this example we define a callback-routine for printing the current estimate for the residual and compare it with the true residual.
 *  To do so, we need to pass the system matrix, the right hand side, and the initial guess to the monitor routine, which we achieve by packing pointers to these objects into a struct:
 **/

template<typename MatrixT, typename VectorT>
struct monitor_user_data
{
  monitor_user_data(MatrixT const & A, VectorT const & b, VectorT const & guess) : A_ptr(&A), b_ptr(&b), guess_ptr(&guess) {}

  MatrixT const *A_ptr;
  VectorT const *b_ptr;
  VectorT const *guess_ptr;
};

/**
 *  The actual callback-routine takes the current approximation to the result as the first parameter, and the current estimate for the relative residual norm as second argument.
 *  The third argument is a pointer to our user-data, which in a first step cast to the correct type.
 *  If the monitor returns true, the iterative solver stops. This is handy for defining custom termination criteria, e.g. one-norms for the result change.
 *  Since we do not want to terminate the iterative solver with a custom criterion here, we always return 'false' at the end of the function.
 *
 *  Note to type-safety evangelists: This void*-interface is designed to be later exposed through a shared library ('libcuarma').
 *  Thus, user types may not be known at the point of compilation, requiring a void*-approach.
 **/
template<typename VectorT, typename NumericT, typename MatrixT>
bool my_custom_monitor(VectorT const & current_approx, NumericT residual_estimate, void *user_data)
{
  // Extract residual:
  monitor_user_data<MatrixT, VectorT> const *data = reinterpret_cast<monitor_user_data<MatrixT, VectorT> const*>(user_data);

  // Form residual r = b - A*x, taking an initial guess into account: r = b - A * (current_approx + x_initial)
  VectorT x = current_approx + *data->guess_ptr;
  VectorT residual = *data->b_ptr - cuarma::blas::prod(*data->A_ptr, x);
  VectorT initial_residual = *data->b_ptr - cuarma::blas::prod(*data->A_ptr, *data->guess_ptr);

  std::cout << "Residual estimate vs. true residual: " << residual_estimate << " vs. " << cuarma::blas::norm_2(residual) / cuarma::blas::norm_2(initial_residual) << std::endl;

  return false; // no termination of iteration
}


int main()
{
  std::vector<std::map<unsigned int, ScalarType> > stl_A;

  if (!cuarma::io::read_matrix_market_file(stl_A, "data/mat65k.mtx"))
  {
    std::cout << "Error reading Matrix file" << std::endl;
    return EXIT_FAILURE;
  }
  cuarma::compressed_matrix<ScalarType> A;
  cuarma::copy(stl_A, A);

  /**
  * Set up right hand side and reference solution consisting of all ones:
  **/
  cuarma::vector<ScalarType> ref_result = cuarma::scalar_vector<ScalarType>(A.size2(), ScalarType(1.0));
  cuarma::vector<ScalarType> result(A.size2());

  cuarma::vector<ScalarType> b = cuarma::blas::prod(A, ref_result);

  cuarma::vector<ScalarType> init_guess = cuarma::scalar_vector<ScalarType>(ref_result.size(), ScalarType(0.9));
  init_guess[0] = 0;

  /**
   * Set up the monitor data, holding the system matrix, the right hand side, and the initial guess:
   **/
  monitor_user_data<cuarma::compressed_matrix<ScalarType>, cuarma::vector<ScalarType> > my_monitor_data(A, b, init_guess);


  /**
  * set up Jacobi preconditioners (just for demonstration purposes, can be any other preconditioner here):
  **/
  cuarma::blas::jacobi_precond< cuarma::compressed_matrix<ScalarType> > jacobi(A, cuarma::blas::jacobi_tag());


  /**
  * <h2>Conjugate Gradient Solver</h2>
  **/
  std::cout << "----- CG Method -----" << std::endl;

  /**
  * Run the CG method with a relative tolerance of 1e-5 and a maximum of 20 iterations.
  * We instantiate the CG solver object, register the monitor callback (with data), set the initial guess, and launch the solver.
  **/
  cuarma::blas::cg_tag my_cg_tag(1e-5, 20);
  cuarma::blas::cg_solver<cuarma::vector<ScalarType> > my_cg_solver(my_cg_tag);

  my_cg_solver.set_monitor(my_custom_monitor<cuarma::vector<ScalarType>, ScalarType, cuarma::compressed_matrix<ScalarType> >, &my_monitor_data);
  my_cg_solver.set_initial_guess(init_guess);

  my_cg_solver(A, b); // without preconditioner


  /**
  * <h2>Stabilized BiConjugate Gradient Solver</h2>
  **/
  std::cout << "----- BiCGStab Method -----" << std::endl;

  /**
  * Run the Jacobi-preconditioned BiCGStab method with a relative tolerance of 1e-5 and a maximum of 20 iterations.
  * We instantiate the BiCGStab solver object, register the monitor callback (with data), set the initial guess, and launch the solver.
  **/
  cuarma::blas::bicgstab_tag my_bicgstab_tag(1e-5, 20);
  cuarma::blas::bicgstab_solver<cuarma::vector<ScalarType> > my_bicgstab_solver(my_bicgstab_tag);

  my_bicgstab_solver.set_monitor(my_custom_monitor<cuarma::vector<ScalarType>, ScalarType, cuarma::compressed_matrix<ScalarType> >, &my_monitor_data);
  my_bicgstab_solver.set_initial_guess(init_guess);

  my_bicgstab_solver(A, b, jacobi); // with Jacobi preconditioner


  /**
  * <h2>GMRES Solver</h2>
  **/
  std::cout << "----- GMRES Method -----" << std::endl;

  /**
  * Run the unpreconditioned GMRES method with a relative tolerance of 1e-5 and a maximum of 30 iterations for a Krylov size of 10 (i.e. restart every 10 iterations).
  * We instantiate the GMRES solver object, register the monitor callback (with data), set the initial guess, and launch the solver.
  *
  * Note that the monitor in the GMRES method is only called after each restart, but not in every (inner) iteration.
  **/
  cuarma::blas::gmres_tag my_gmres_tag(1e-5, 30, 10);
  cuarma::blas::gmres_solver<cuarma::vector<ScalarType> > my_gmres_solver(my_gmres_tag);

  my_gmres_solver.set_monitor(my_custom_monitor<cuarma::vector<ScalarType>, ScalarType, cuarma::compressed_matrix<ScalarType> >, &my_monitor_data);
  my_gmres_solver.set_initial_guess(init_guess);

  my_gmres_solver(A, b);

  /**
  *  That's it, the tutorial is completed.
  **/
  std::cout << "!!!! TUTORIAL COMPLETED SUCCESSFULLY !!!!" << std::endl;

  return EXIT_SUCCESS;
}

