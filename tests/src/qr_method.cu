#include "hip/hip_runtime.h"
/* =========================================================================
      Copyright (c) 2015-2017, COE of Peking University, Shaoqiang Tang.

                         -----------------
            cuarma - COE of Peking University, Shaoqiang Tang.
                         -----------------

                  Author Email    yangxianpku@pku.edu.cn

         Code Repo   https://github.com/yangxianpku/cuarma

                      License:    MIT (X11) License
============================================================================= */

/**  @file   qr_method.cu
 *   @coding UTF-8
 *   @brief  Tests the eigenvalue routines based on the QR method.
 *   @brief  ���ԣ�QR�ֽ��������ֵ
 */

#include <iostream>
#include <fstream>
#include <stdexcept>
#include <vector>

#include "head_define.h"

#include "cuarma/blas/prod.hpp"
#include "cuarma/blas/qr-method.hpp"
#include "cuarma/tools/timer.hpp"
#include <boost/numeric/ublas/vector.hpp>
#include <boost/numeric/ublas/matrix.hpp>

namespace ublas = boost::numeric::ublas;

void read_matrix_size(std::fstream& f, std::size_t& sz);

void read_matrix_size(std::fstream& f, std::size_t& sz)
{
    if(!f.is_open())
    {
        throw std::invalid_argument("File is not opened");
    }

    f >> sz;
}

template <typename NumericT, typename MatrixLayout>
void read_matrix_body(std::fstream& f, cuarma::matrix<NumericT, MatrixLayout>& A)
{
    if(!f.is_open())
    {
        throw std::invalid_argument("File is not opened");
    }

    boost::numeric::ublas::matrix<NumericT> h_A(A.size1(), A.size2());

    for(std::size_t i = 0; i < h_A.size1(); i++) {
        for(std::size_t j = 0; j < h_A.size2(); j++) {
            NumericT val = 0.0;
            f >> val;
            h_A(i, j) = val;
        }
    }

    cuarma::copy(h_A, A);
}

template<typename NumericT>
void read_vector_body(std::fstream& f, std::vector<NumericT>& v) {
    if(!f.is_open())
        throw std::invalid_argument("File is not opened");

    for(std::size_t i = 0; i < v.size(); i++)
    {
            NumericT val = 0.0;
            f >> val;
            v[i] = val;
    }
}

template<typename NumericT, typename MatrixLayout>
bool check_tridiag(cuarma::matrix<NumericT, MatrixLayout>& A_orig, NumericT EPS)
{
    ublas::matrix<NumericT> A(A_orig.size1(), A_orig.size2());
    cuarma::copy(A_orig, A);

    for (unsigned int i = 0; i < A.size1(); i++) {
        for (unsigned int j = 0; j < A.size2(); j++) {
            if ((std::abs(A(i, j)) > EPS) && ((i - 1) != j) && (i != j) && ((i + 1) != j))
            {
                // std::cout << "Failed at " << i << " " << j << " " << A(i, j) << "\n";
                return false;
            }
        }
    }
    return true;
}

template <typename NumericT, typename MatrixLayout>
bool check_hessenberg(cuarma::matrix<NumericT, MatrixLayout>& A_orig, NumericT EPS)
{
    ublas::matrix<NumericT> A(A_orig.size1(), A_orig.size2());
    cuarma::copy(A_orig, A);

    for (std::size_t i = 0; i < A.size1(); i++) {
        for (std::size_t j = 0; j < A.size2(); j++) {
            if ((std::abs(A(i, j)) > EPS) && (i > (j + 1)))
            {
                // std::cout << "Failed at " << i << " " << j << " " << A(i, j) << "\n";
                return false;
            }
        }
    }
    return true;
}

template<typename NumericT>
NumericT matrix_compare(ublas::matrix<NumericT>& res,
                        ublas::matrix<NumericT>& ref)
{
    NumericT diff = 0.0;
    NumericT mx = 0.0;

    for(std::size_t i = 0; i < res.size1(); i++)
    {
        for(std::size_t j = 0; j < res.size2(); j++)
        {
            diff = std::max(diff, std::abs(res(i, j) - ref(i, j)));
            mx = std::max(mx, res(i, j));
        }
    }

    return diff / mx;
}

template<typename NumericT>
NumericT vector_compare(std::vector<NumericT> & res,
                        std::vector<NumericT> & ref)
{
    std::sort(ref.begin(), ref.end());
    std::sort(res.begin(), res.end());

    NumericT diff = 0.0;
    NumericT mx = 0.0;
    for(size_t i = 0; i < res.size(); i++)
    {
        diff = std::max(diff, std::abs(res[i] - ref[i]));
        mx = std::max(mx, res[i]);
    }

    return diff / mx;
}

template <typename NumericT, typename MatrixLayout>
void matrix_print(cuarma::matrix<NumericT, MatrixLayout>& A)
{
    for (unsigned int i = 0; i < A.size1(); i++) {
        for (unsigned int j = 0; j < A.size2(); j++)
           std::cout << std::fixed << A(i, j) << "\t";
        std::cout << "\n";
    }
}

template <typename NumericT, typename MatrixLayout>
void test_eigen(const std::string& fn, bool is_symm, NumericT EPS)
{
    std::cout << "Reading..." << "\n";
    std::size_t sz;
    // read file
    std::fstream f(fn.c_str(), std::fstream::in);
    //read size of input matrix
    read_matrix_size(f, sz);

    bool is_row = cuarma::is_row_major<MatrixLayout>::value;
    if (is_row)
      std::cout << "Testing row-major matrix of size " << sz << "-by-" << sz << std::endl;
    else
      std::cout << "Testing column-major matrix of size " << sz << "-by-" << sz << std::endl;

    cuarma::matrix<NumericT> A_input(sz, sz), A_ref(sz, sz), Q(sz, sz);
    // reference vector with reference values from file
    std::vector<NumericT> eigen_ref_re(sz);
    // calculated real eigenvalues
    std::vector<NumericT> eigen_re(sz);
    // calculated im. eigenvalues
    std::vector<NumericT> eigen_im(sz);

    // read input matrix from file
    read_matrix_body(f, A_input);
    // read reference eigenvalues from file
    read_vector_body(f, eigen_ref_re);


    f.close();

    A_ref = A_input;

    std::cout << "Calculation..." << "\n";

    cuarma::tools::timer timer;
    timer.start();
    // Start the calculation
    if(is_symm)
        cuarma::blas::qr_method_sym(A_input, Q, eigen_re);
    else
        cuarma::blas::qr_method_nsm(A_input, Q, eigen_re, eigen_im);
/*

    std::cout << "\n\n Matrix A: \n\n";
    matrix_print(A_input);
    std::cout << "\n\n";

    std::cout << "\n\n Matrix Q: \n\n";
    matrix_print(Q);
    std::cout << "\n\n";
*/

    double time_spend = timer.get();

    std::cout << "Verification..." << "\n";

    bool is_hessenberg = check_hessenberg(A_input, EPS);
    bool is_tridiag = check_tridiag(A_input, EPS);

    ublas::matrix<NumericT> A_ref_ublas(sz, sz), A_input_ublas(sz, sz), Q_ublas(sz, sz), result1(sz, sz), result2(sz, sz);
    cuarma::copy(A_ref, A_ref_ublas);
    cuarma::copy(A_input, A_input_ublas);
    cuarma::copy(Q, Q_ublas);

    // compute result1 = ublas::prod(Q_ublas, A_input_ublas);   (terribly slow when using ublas directly)
    for (std::size_t i=0; i<result1.size1(); ++i)
      for (std::size_t j=0; j<result1.size2(); ++j)
      {
        NumericT value = 0;
        for (std::size_t k=0; k<Q_ublas.size2(); ++k)
          value += Q_ublas(i, k) * A_input_ublas(k, j);
        result1(i,j) = value;
      }
    // compute result2 = ublas::prod(A_ref_ublas, Q_ublas);   (terribly slow when using ublas directly)
    for (std::size_t i=0; i<result2.size1(); ++i)
      for (std::size_t j=0; j<result2.size2(); ++j)
      {
        NumericT value = 0;
        for (std::size_t k=0; k<A_ref_ublas.size2(); ++k)
          value += A_ref_ublas(i, k) * Q_ublas(k, j);
        result2(i,j) = value;
      }


    NumericT prods_diff = matrix_compare(result1, result2);
    NumericT eigen_diff = vector_compare(eigen_re, eigen_ref_re);


    bool is_ok = is_hessenberg;

    if(is_symm)
        is_ok = is_ok && is_tridiag;

    is_ok = is_ok && (eigen_diff < EPS);
    is_ok = is_ok && (prods_diff < EPS);

    // std::cout << A_ref << "\n";
    // std::cout << A_input << "\n";
    // std::cout << Q << "\n";
    // std::cout << eigen_re << "\n";
    // std::cout << eigen_im << "\n";
    // std::cout << eigen_ref_re << "\n";
    // std::cout << eigen_ref_im << "\n";

    // std::cout << result1 << "\n";
    // std::cout << result2 << "\n";
    // std::cout << eigen_ref << "\n";
    // std::cout << eigen << "\n";

    printf("%6s [%dx%d] %40s time = %.4f\n", is_ok?"[[OK]]":"[FAIL]", (int)A_ref.size1(), (int)A_ref.size2(), fn.c_str(), time_spend);
    printf("tridiagonal = %d, hessenberg = %d prod-diff = %f eigen-diff = %f\n", is_tridiag, is_hessenberg, prods_diff, eigen_diff);
    std::cout << std::endl << std::endl;

    if (!is_ok)
      exit(EXIT_FAILURE);

}

int main()
{
  float epsilon1 = 0.0001f;

  std::cout << "# Testing setup:" << std::endl;
  std::cout << "  eps:     " << epsilon1 << std::endl;
  std::cout << "  numeric: double" << std::endl;
  std::cout << std::endl;
  test_eigen<float, cuarma::row_major   >("../examples/testdata/eigen/symm5.example", true, epsilon1);
  test_eigen<float, cuarma::column_major>("../examples/testdata/eigen/symm5.example", true, epsilon1);

  {
    double epsilon2 = 1e-5;

    std::cout << "# Testing setup:" << std::endl;
    std::cout << "  eps:     " << epsilon2 << std::endl;
    std::cout << "  numeric: double" << std::endl;
    std::cout << std::endl;
    test_eigen<double, cuarma::row_major   >("../examples/testdata/eigen/symm5.example", true, epsilon2);
    test_eigen<double, cuarma::column_major>("../examples/testdata/eigen/symm5.example", true, epsilon2);
  }

  //test_eigen<cuarma::row_major>("../../examples/testdata/eigen/symm3.example", true);  // Computation of this matrix takes very long
  //test_eigen<cuarma::column_major>("../../examples/testdata/eigen/symm3.example", true);

  //test_eigen<cuarma::row_major>("../examples/testdata/eigen/nsm2.example", false);
  //test_eigen<cuarma::row_major>("../../examples/testdata/eigen/nsm2.example", false);
  //test_eigen("../../examples/testdata/eigen/nsm3.example", false);
  //test_eigen("../../examples/testdata/eigen/nsm4.example", false); //Note: This test suffers from round-off errors in single precision, hence disabled

  std::cout << std::endl;
  std::cout << "------- Test completed --------" << std::endl;
  std::cout << std::endl;

  return EXIT_SUCCESS;
}
