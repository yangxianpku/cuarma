#include "hip/hip_runtime.h"
/* =========================================================================
      Copyright (c) 2015-2017, COE of Peking University, Shaoqiang Tang.

                         -----------------
            cuarma - COE of Peking University, Shaoqiang Tang.
                         -----------------

                  Author Email    yangxianpku@pku.edu.cn

         Code Repo   https://github.com/yangxianpku/cuarma

                      License:    MIT (X11) License
============================================================================= */

/**  @file   bisect.cu
 *   @coding UTF-8
 *   @brief  Computation of eigenvalues of a symmetric, tridiagonal matrix using bisection.
 *   @brief  Tests the bisection implementation for symmetric tridiagonal matrices.
 *   @brief  ���ԣ����ַ�����Գơ����Ǿ��������ֵ
 */

#include <stdlib.h>
#include <stdio.h>
#include <string.h>

#include "head_define.h"

#include "cuarma/scalar.hpp"
#include "cuarma/vector.hpp"

#include "cuarma/blas/bisect.hpp"
#include "cuarma/blas/bisect_gpu.hpp"
#include "cuarma/blas/tql2.hpp"

#define EPS 10.0e-4

typedef float NumericT;

bool runTest(unsigned int mat_size);

////////////////////////////////////////////////////////////////////////////////
/// \brief initInputData   Initialize the diagonal and superdiagonal elements of
///                        the matrix
/// \param diagonal        diagonal elements of the matrix
/// \param superdiagonal   superdiagonal elements of the matrix
/// \param mat_size        Dimension of the matrix
///
template<typename NumericT>
void initInputData(std::vector<NumericT> &diagonal, std::vector<NumericT> &superdiagonal, unsigned int mat_size)
{
 
  srand(278217421);

#define RANDOM_VALUES false

  if (RANDOM_VALUES == true)
  {
    // Initialize diagonal and superdiagonal elements with random values
    for (unsigned int i = 0; i < mat_size; ++i)
    {
        diagonal[i] =      static_cast<NumericT>(2.0 * (((double)rand() / (double) RAND_MAX) - 0.5));
        superdiagonal[i] = static_cast<NumericT>(2.0 * (((double)rand() / (double) RAND_MAX) - 0.5));
    }
  }
  else
  { 
    // Initialize diagonal and superdiagonal elements with modulo values
    // This will cause in many multiple eigenvalues.
    for (unsigned int i = 0; i < mat_size; ++i)
    {
       diagonal[i] = ((NumericT)(i % 3)) - 4.5f;
       superdiagonal[i] = ((NumericT)(i % 3)) - 5.5f;
    }
  }
  // the first element of s is used as padding on the device (thus the
  // whole vector is copied to the device but the kernels are launched
  // with (s+1) as start address
  superdiagonal[0] = 0.0f; 
}

int main()
{
    bool test_result = false;

    // run test for large matrix
    test_result = runTest(550);
    if(test_result == true)
    {
      std::cout << "First Test Succeeded!" << std::endl << std::endl;
    }
    else
    {
      std::cout << "---TEST FAILED---" << std::endl;
      return EXIT_FAILURE;
    }

    // run test for small matrix
    test_result = runTest(96);

    if(test_result == true)
   {
      std::cout << std::endl << "---TEST SUCCESSFULLY COMPLETED---" << std::endl;
      return EXIT_SUCCESS;
    }
    else
    {
      std::cout << "---TEST FAILED---" << std::endl;
      return EXIT_FAILURE;
    }
}

bool runTest(unsigned int mat_size)
{
    bool bResult = false;

    std::vector<NumericT> diagonal(mat_size);
    std::vector<NumericT> superdiagonal(mat_size);
    std::vector<NumericT> eigenvalues_bisect(mat_size);

    // -------------Initialize data-------------------
    // Fill the diagonal and superdiagonal elements of the vector
    initInputData(diagonal, superdiagonal, mat_size);

    // -------Start the bisection algorithm------------
    std::cout << "Start the bisection algorithm" << std::endl;
    std::cout << "Matrix size: " << mat_size << std::endl;
    bResult = cuarma::blas::bisect(diagonal, superdiagonal, eigenvalues_bisect);

    // Exit if an error occured during the execution of the algorithm
    if (bResult == false)
     return false;

    // ---------------Check the results---------------
    // The results of the bisection algorithm will be checked with the tql algorithm
    // Initialize Data for tql1 algorithm

    std::vector<NumericT> diagonal_tql(mat_size);
    std::vector<NumericT> superdiagonal_tql(mat_size);
    diagonal_tql = diagonal;
    superdiagonal_tql = superdiagonal;

    // Start the tql algorithm
    std::cout << "Start the tql algorithm..." << std::endl;
    cuarma::blas::tql1<NumericT>(mat_size, diagonal_tql, superdiagonal_tql);

    // Ensure that eigenvalues from tql1 algorithm are sorted in ascending order
    std::sort(diagonal_tql.begin(), diagonal_tql.end());

    // Compare the results from the bisection algorithm with the results
    // from the tql algorithm.
    std::cout << "Start comparison..." << std::endl;
    for (unsigned int i = 0; i < mat_size; i++)
    {
       if (std::abs(diagonal_tql[i] - eigenvalues_bisect[i]) > EPS)
       {
         std::cout << std::setprecision(12) << diagonal_tql[i] << "  != " << eigenvalues_bisect[i] << "\n";
         return false;
       }
    }

  return bResult;
    
}
