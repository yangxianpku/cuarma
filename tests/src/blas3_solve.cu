#include "hip/hip_runtime.h"
/* =========================================================================
      Copyright (c) 2015-2017, COE of Peking University, Shaoqiang Tang.

                         -----------------
            cuarma - COE of Peking University, Shaoqiang Tang.
                         -----------------

                  Author Email    yangxianpku@pku.edu.cn

         Code Repo   https://github.com/yangxianpku/cuarma

                      License:    MIT (X11) License
============================================================================= */

/**  @file   blas3_solve.cu
 *   @coding UTF-8
 *   @brief  ests the BLAS level 3 triangular solvers.
 *   @brief  ���ԣ�BLAS level 3 triangular solvers
 */

#include <iostream>
#include <vector>

#include "head_define.h"

#include "cuarma/scalar.hpp"
#include "cuarma/matrix.hpp"
#include "cuarma/matrix_proxy.hpp"
#include "cuarma/vector.hpp"
#include "cuarma/blas/prod.hpp"
#include "cuarma/blas/norm_2.hpp"
#include "cuarma/blas/direct_solve.hpp"
#include "cuarma/tools/random.hpp"

template<typename ScalarType>
ScalarType diff(ScalarType & s1, cuarma::scalar<ScalarType> & s2)
{
   cuarma::backend::finish();
   if (s1 != s2)
      return (s1 - s2) / std::max(fabs(s1), fabs(s2));
   return 0;
}

template<typename ScalarType>
ScalarType diff(std::vector<ScalarType> & v1, cuarma::vector<ScalarType> & v2)
{
   std::vector<ScalarType> v2_cpu(v2.size());
   cuarma::backend::finish();
   cuarma::copy(v2.begin(), v2.end(), v2_cpu.begin());
   cuarma::backend::finish();

   for (std::size_t i=0;i<v1.size(); ++i)
   {
      if ( std::max( fabs(v2_cpu[i]), fabs(v1[i]) ) > 0 )
         v2_cpu[i] = fabs(v2_cpu[i] - v1[i]) / std::max( fabs(v2_cpu[i]), fabs(v1[i]) );
      else
         v2_cpu[i] = 0.0;
   }

   return norm_inf(v2_cpu);
}


template<typename ScalarType, typename VCLMatrixType>
ScalarType diff(std::vector<std::vector<ScalarType> > & mat1, VCLMatrixType & mat2)
{
   std::vector<std::vector<ScalarType> > mat2_cpu(mat2.size1(), std::vector<ScalarType>(mat2.size2()));
   cuarma::backend::finish();  //workaround for a bug in APP SDK 2.7 on Trinity APUs (with Catalyst 12.8)
   cuarma::copy(mat2, mat2_cpu);
   ScalarType ret = 0;
   ScalarType act = 0;

    for (unsigned int i = 0; i < mat2_cpu.size(); ++i)
    {
      for (unsigned int j = 0; j < mat2_cpu[i].size(); ++j)
      {
        act = std::fabs(mat2_cpu[i][j] - mat1[i][j]) / std::max( std::fabs(mat2_cpu[i][j]), std::fabs(mat1[i][j]) );
         if (act > ret)
           ret = act;
      }
    }
   //std::cout << ret << std::endl;
   return ret;
}


// Triangular solvers
template<typename NumericT>
void inplace_solve_lower(std::vector<std::vector<NumericT> > const & A, std::vector<std::vector<NumericT> > & B, bool unit_diagonal)
{
  for (std::size_t i=0; i<A.size(); ++i)
  {
    for (std::size_t j=0; j < i; ++j)
    {
      NumericT val_A = A[i][j];
      for (std::size_t k=0; k<B[i].size(); ++k)
        B[i][k] -= val_A * B[j][k];
    }

    NumericT diag_A = unit_diagonal ? NumericT(1) : A[i][i];

    for (std::size_t k=0; k<B[i].size(); ++k)
      B[i][k] /= diag_A;
  }
}

template<typename NumericT>
void inplace_solve(std::vector<std::vector<NumericT> > const & A, std::vector<std::vector<NumericT> > & B, cuarma::blas::lower_tag)
{
  inplace_solve_lower(A, B, false);
}

template<typename NumericT>
void inplace_solve(std::vector<std::vector<NumericT> > const & A, std::vector<std::vector<NumericT> > & B, cuarma::blas::unit_lower_tag)
{
  inplace_solve_lower(A, B, true);
}

template<typename NumericT>
void inplace_solve_upper(std::vector<std::vector<NumericT> > const & A, std::vector<std::vector<NumericT> > & B, bool unit_diagonal)
{
  for (std::size_t i2=0; i2<A.size(); ++i2)
  {
    std::size_t i = A.size() - i2 - 1;
    for (std::size_t j=i+1; j < A[0].size(); ++j)
    {
      NumericT val_A = A[i][j];
      for (std::size_t k=0; k<B[i].size(); ++k)
        B[i][k] -= val_A * B[j][k];
    }

    NumericT diag_A = unit_diagonal ? NumericT(1) : A[i][i];

    for (std::size_t k=0; k<B[i].size(); ++k)
      B[i][k] /= diag_A;
  }
}

template<typename NumericT>
void inplace_solve(std::vector<std::vector<NumericT> > const & A, std::vector<std::vector<NumericT> > & B, cuarma::blas::upper_tag)
{
  inplace_solve_upper(A, B, false);
}

template<typename NumericT>
void inplace_solve(std::vector<std::vector<NumericT> > const & A, std::vector<std::vector<NumericT> > & B, cuarma::blas::unit_upper_tag)
{
  inplace_solve_upper(A, B, true);
}

template<typename NumericT, typename SolverTagT>
std::vector<std::vector<NumericT> > solve(std::vector<std::vector<NumericT> > const & A, std::vector<std::vector<NumericT> > const & B, SolverTagT)
{
  std::vector<std::vector<NumericT> > C(B);
  inplace_solve(A, C, SolverTagT());
  return C;
}


template<typename RHSTypeRef, typename RHSTypeCheck, typename Epsilon >
void run_solver_check(RHSTypeRef & B_ref, RHSTypeCheck & B_check, int & retval, Epsilon const & epsilon)
{
   double act_diff = fabs(diff(B_ref, B_check));
   if ( act_diff > epsilon )
   {
     std::cout << " FAILED!" << std::endl;
     std::cout << "# Error at operation: matrix-matrix solve" << std::endl;
     std::cout << "  diff: " << act_diff << std::endl;
     retval = EXIT_FAILURE;
   }
   else
     std::cout << " passed! " << act_diff << std::endl;

}

template<typename NumericT>
std::vector<std::vector<NumericT> > trans(std::vector<std::vector<NumericT> > const & A)
{
  std::vector<std::vector<NumericT> > A_trans(A[0].size(), std::vector<NumericT>(A.size()));

  for (std::size_t i=0; i<A.size(); ++i)
    for (std::size_t j=0; j<A[i].size(); ++j)
      A_trans[j][i] = A[i][j];

  return A_trans;
}


template< typename NumericT, typename Epsilon, typename ReferenceMatrixTypeA, typename ReferenceMatrixTypeB, 
	typename ReferenceMatrixTypeC,typename MatrixTypeA, typename MatrixTypeB, typename MatrixTypeC, typename MatrixTypeResult>
int test_solve(Epsilon const& epsilon,ReferenceMatrixTypeA const & A,ReferenceMatrixTypeB const & B_start,
              ReferenceMatrixTypeC const & C_start, MatrixTypeA const & arma_A, MatrixTypeB & arma_B, MatrixTypeC & arma_C, MatrixTypeResult const & )
{
   int retval = EXIT_SUCCESS;

   // --------------------------------------------------------------------------

   ReferenceMatrixTypeA result;
   ReferenceMatrixTypeC C_trans;

   ReferenceMatrixTypeB B = B_start;
   ReferenceMatrixTypeC C = C_start;

   MatrixTypeResult arma_result;

   // Test: A \ B with various tags --------------------------------------------------------------------------
   std::cout << "Testing A \\ B: " << std::endl;
   std::cout << " * upper_tag:      ";
   result = solve(A, B, cuarma::blas::upper_tag());
   arma_result = cuarma::blas::solve(arma_A, arma_B, cuarma::blas::upper_tag());
   run_solver_check(result, arma_result, retval, epsilon);

   std::cout << " * unit_upper_tag: ";
   result = solve(A, B, cuarma::blas::unit_upper_tag());
   arma_result = cuarma::blas::solve(arma_A, arma_B, cuarma::blas::unit_upper_tag());
   run_solver_check(result, arma_result, retval, epsilon);

   std::cout << " * lower_tag:      ";
   result = solve(A, B, cuarma::blas::lower_tag());
   arma_result = cuarma::blas::solve(arma_A, arma_B, cuarma::blas::lower_tag());
   run_solver_check(result, arma_result, retval, epsilon);

   std::cout << " * unit_lower_tag: ";
   result = solve(A, B, cuarma::blas::unit_lower_tag());
   arma_result = cuarma::blas::solve(arma_A, arma_B, cuarma::blas::unit_lower_tag());
   run_solver_check(result, arma_result, retval, epsilon);

   if (retval == EXIT_SUCCESS)
     std::cout << "Test A \\ B passed!" << std::endl;

   B = B_start;
   C = C_start;

   // Test: A \ B^T --------------------------------------------------------------------------
   std::cout << "Testing A \\ B^T: " << std::endl;
   std::cout << " * upper_tag:      ";
   cuarma::copy(C, arma_C); C_trans = trans(C);

   //check solve():
   result = solve(A, C_trans, cuarma::blas::upper_tag());
   arma_result = cuarma::blas::solve(arma_A, trans(arma_C), cuarma::blas::upper_tag());
   run_solver_check(result, arma_result, retval, epsilon);

   //check compute kernels:
   std::cout << " * upper_tag:      ";
   inplace_solve(A, C_trans, cuarma::blas::upper_tag());
   cuarma::blas::inplace_solve(arma_A, trans(arma_C), cuarma::blas::upper_tag());
   C = trans(C_trans); run_solver_check(C, arma_C, retval, epsilon);

   std::cout << " * unit_upper_tag: ";
   cuarma::copy(C, arma_C); C_trans = trans(C);
   inplace_solve(A, C_trans, cuarma::blas::unit_upper_tag());
   cuarma::blas::inplace_solve(arma_A, trans(arma_C), cuarma::blas::unit_upper_tag());
   C = trans(C_trans); run_solver_check(C, arma_C, retval, epsilon);

   std::cout << " * lower_tag:      ";
   cuarma::copy(C, arma_C); C_trans = trans(C);
   inplace_solve(A, C_trans, cuarma::blas::lower_tag());
   cuarma::blas::inplace_solve(arma_A, trans(arma_C), cuarma::blas::lower_tag());
   C = trans(C_trans); run_solver_check(C, arma_C, retval, epsilon);

   std::cout << " * unit_lower_tag: ";
   cuarma::copy(C, arma_C); C_trans = trans(C);
   inplace_solve(A, C_trans, cuarma::blas::unit_lower_tag());
   cuarma::blas::inplace_solve(arma_A, trans(arma_C), cuarma::blas::unit_lower_tag());
   C = trans(C_trans); run_solver_check(C, arma_C, retval, epsilon);

   if (retval == EXIT_SUCCESS)
     std::cout << "Test A \\ B^T passed!" << std::endl;

   B = B_start;
   C = C_start;

   // Test: A \ B with various tags --------------------------------------------------------------------------
   std::cout << "Testing A^T \\ B: " << std::endl;
   std::cout << " * upper_tag:      ";
   cuarma::copy(B, arma_B);
   result = solve(trans(A), B, cuarma::blas::upper_tag());
   arma_result = cuarma::blas::solve(trans(arma_A), arma_B, cuarma::blas::upper_tag());
   run_solver_check(result, arma_result, retval, epsilon);

   std::cout << " * unit_upper_tag: ";
   cuarma::copy(B, arma_B);
   result = solve(trans(A), B, cuarma::blas::unit_upper_tag());
   arma_result = cuarma::blas::solve(trans(arma_A), arma_B, cuarma::blas::unit_upper_tag());
   run_solver_check(result, arma_result, retval, epsilon);

   std::cout << " * lower_tag:      ";
   cuarma::copy(B, arma_B);
   result = solve(trans(A), B, cuarma::blas::lower_tag());
   arma_result = cuarma::blas::solve(trans(arma_A), arma_B, cuarma::blas::lower_tag());
   run_solver_check(result, arma_result, retval, epsilon);

   std::cout << " * unit_lower_tag: ";
   cuarma::copy(B, arma_B);
   result = solve(trans(A), B, cuarma::blas::unit_lower_tag());
   arma_result = cuarma::blas::solve(trans(arma_A), arma_B, cuarma::blas::unit_lower_tag());
   run_solver_check(result, arma_result, retval, epsilon);

   if (retval == EXIT_SUCCESS)
     std::cout << "Test A^T \\ B passed!" << std::endl;

   B = B_start;
   C = C_start;

   // Test: A^T \ B^T --------------------------------------------------------------------------
   std::cout << "Testing A^T \\ B^T: " << std::endl;
   std::cout << " * upper_tag:      ";
   cuarma::copy(C, arma_C); C_trans = trans(C);
   //check solve():
   result = solve(trans(A), C_trans, cuarma::blas::upper_tag());
   arma_result = cuarma::blas::solve(trans(arma_A), trans(arma_C), cuarma::blas::upper_tag());
   run_solver_check(result, arma_result, retval, epsilon);
   //check kernels:
   std::cout << " * upper_tag:      ";
   inplace_solve(trans(A), C_trans, cuarma::blas::upper_tag());
   cuarma::blas::inplace_solve(trans(arma_A), trans(arma_C), cuarma::blas::upper_tag());
   C = trans(C_trans); run_solver_check(C, arma_C, retval, epsilon);

   std::cout << " * unit_upper_tag: ";
   cuarma::copy(C, arma_C); C_trans = trans(C);
   inplace_solve(trans(A), C_trans, cuarma::blas::unit_upper_tag());
   cuarma::blas::inplace_solve(trans(arma_A), trans(arma_C), cuarma::blas::unit_upper_tag());
   C = trans(C_trans); run_solver_check(C, arma_C, retval, epsilon);

   std::cout << " * lower_tag:      ";
   cuarma::copy(C, arma_C); C_trans = trans(C);
   inplace_solve(trans(A), C_trans, cuarma::blas::lower_tag());
   cuarma::blas::inplace_solve(trans(arma_A), trans(arma_C), cuarma::blas::lower_tag());
   C = trans(C_trans); run_solver_check(C, arma_C, retval, epsilon);

   std::cout << " * unit_lower_tag: ";
   cuarma::copy(C, arma_C); C_trans = trans(C);
   inplace_solve(trans(A), C_trans, cuarma::blas::unit_lower_tag());
   cuarma::blas::inplace_solve(trans(arma_A), trans(arma_C), cuarma::blas::unit_lower_tag());
   C = trans(C_trans); run_solver_check(C, arma_C, retval, epsilon);

   if (retval == EXIT_SUCCESS)
     std::cout << "Test A^T \\ B^T passed!" << std::endl;

   return retval;
}


template< typename NumericT, typename F_A, typename F_B, typename Epsilon >
int test_solve(Epsilon const& epsilon)
{
  cuarma::tools::uniform_random_numbers<NumericT> randomNumber;

  int ret = EXIT_SUCCESS;
  std::size_t matrix_size = 135;  //some odd number, not too large
  std::size_t rhs_num = 67;

  std::cout << "--- Part 2: Testing matrix-matrix solver ---" << std::endl;


  std::vector<std::vector<NumericT> > A(matrix_size, std::vector<NumericT>(matrix_size));
  std::vector<std::vector<NumericT> > B_start(matrix_size,  std::vector<NumericT>(rhs_num));
  std::vector<std::vector<NumericT> > C_start(rhs_num,  std::vector<NumericT>(matrix_size));

  for (std::size_t i = 0; i < A.size(); ++i)
  {
    for (std::size_t j = 0; j < A[i].size(); ++j)
        A[i][j] = static_cast<NumericT>(-0.5) * randomNumber();
    A[i][i] = NumericT(1.0) + NumericT(2.0) * randomNumber(); //some extra weight on diagonal for stability
  }

  for (std::size_t i = 0; i < B_start.size(); ++i)
    for (std::size_t j = 0; j < B_start[i].size(); ++j)
      B_start[i][j] = randomNumber();

  for (std::size_t i = 0; i < C_start.size(); ++i)
    for (std::size_t j = 0; j < C_start[i].size(); ++j)
      C_start[i][j] = randomNumber();


  // A
  cuarma::range range1_A(matrix_size, 2*matrix_size);
  cuarma::range range2_A(2*matrix_size, 3*matrix_size);
  cuarma::slice slice1_A(matrix_size, 2, matrix_size);
  cuarma::slice slice2_A(0, 3, matrix_size);

  cuarma::matrix<NumericT, F_A>    arma_A(matrix_size, matrix_size);
  cuarma::copy(A, arma_A);

  cuarma::matrix<NumericT, F_A>    arma_big_range_A(4*matrix_size, 4*matrix_size);
  cuarma::matrix_range<cuarma::matrix<NumericT, F_A> > arma_range_A(arma_big_range_A, range1_A, range2_A);
  cuarma::copy(A, arma_range_A);

  cuarma::matrix<NumericT, F_A>    arma_big_slice_A(4*matrix_size, 4*matrix_size);
  cuarma::matrix_slice<cuarma::matrix<NumericT, F_A> > arma_slice_A(arma_big_slice_A, slice1_A, slice2_A);
  cuarma::copy(A, arma_slice_A);


  // B
  cuarma::range range1_B(matrix_size, 2*matrix_size);
  cuarma::range range2_B(2*rhs_num, 3*rhs_num);
  cuarma::slice slice1_B(matrix_size, 2, matrix_size);
  cuarma::slice slice2_B(0, 3, rhs_num);

  cuarma::matrix<NumericT, F_B>    arma_B(matrix_size, rhs_num);
  cuarma::copy(B_start, arma_B);

  cuarma::matrix<NumericT, F_B>    arma_big_range_B(4*matrix_size, 4*rhs_num);
  cuarma::matrix_range<cuarma::matrix<NumericT, F_B> > arma_range_B(arma_big_range_B, range1_B, range2_B);
  cuarma::copy(B_start, arma_range_B);

  cuarma::matrix<NumericT, F_B>    arma_big_slice_B(4*matrix_size, 4*rhs_num);
  cuarma::matrix_slice<cuarma::matrix<NumericT, F_B> > arma_slice_B(arma_big_slice_B, slice1_B, slice2_B);
  cuarma::copy(B_start, arma_slice_B);


  // C
  cuarma::range range1_C(rhs_num, 2*rhs_num);
  cuarma::range range2_C(2*matrix_size, 3*matrix_size);
  cuarma::slice slice1_C(rhs_num, 2, rhs_num);
  cuarma::slice slice2_C(0, 3, matrix_size);

  cuarma::matrix<NumericT, F_B>    arma_C(rhs_num, matrix_size);
  cuarma::copy(C_start, arma_C);

  cuarma::matrix<NumericT, F_B>    arma_big_range_C(4*rhs_num, 4*matrix_size);
  cuarma::matrix_range<cuarma::matrix<NumericT, F_B> > arma_range_C(arma_big_range_C, range1_C, range2_C);
  cuarma::copy(C_start, arma_range_C);

  cuarma::matrix<NumericT, F_B>    arma_big_slice_C(4*rhs_num, 4*matrix_size);
  cuarma::matrix_slice<cuarma::matrix<NumericT, F_B> > arma_slice_C(arma_big_slice_C, slice1_C, slice2_C);
  cuarma::copy(C_start, arma_slice_C);


  std::cout << "Now using A=matrix, B=matrix" << std::endl;
  ret = test_solve<NumericT>(epsilon,
                             A, B_start, C_start,
                             arma_A, arma_B, arma_C, arma_B
                            );
  if (ret != EXIT_SUCCESS)
    return ret;

  std::cout << "Now using A=matrix, B=range" << std::endl;
  ret = test_solve<NumericT>(epsilon,
                             A, B_start, C_start,
                             arma_A, arma_range_B, arma_range_C, arma_B
                            );
  if (ret != EXIT_SUCCESS)
    return ret;

  std::cout << "Now using A=matrix, B=slice" << std::endl;
  ret = test_solve<NumericT>(epsilon,
                             A, B_start, C_start,
                             arma_A, arma_slice_B, arma_slice_C, arma_B
                            );
  if (ret != EXIT_SUCCESS)
    return ret;



  std::cout << "Now using A=range, B=matrix" << std::endl;
  ret = test_solve<NumericT>(epsilon,
                             A, B_start, C_start,
                             arma_range_A, arma_B, arma_C, arma_B
                            );
  if (ret != EXIT_SUCCESS)
    return ret;

  std::cout << "Now using A=range, B=range" << std::endl;
  ret = test_solve<NumericT>(epsilon,
                             A, B_start, C_start,
                             arma_range_A, arma_range_B, arma_range_C, arma_B
                            );
  if (ret != EXIT_SUCCESS)
    return ret;

  std::cout << "Now using A=range, B=slice" << std::endl;
  ret = test_solve<NumericT>(epsilon,
                             A, B_start, C_start,
                             arma_range_A, arma_slice_B, arma_slice_C, arma_B
                            );
  if (ret != EXIT_SUCCESS)
    return ret;


  std::cout << "Now using A=slice, B=matrix" << std::endl;
  ret = test_solve<NumericT>(epsilon,
                             A, B_start, C_start,
                             arma_slice_A, arma_B, arma_C, arma_B
                            );
  if (ret != EXIT_SUCCESS)
    return ret;

  std::cout << "Now using A=slice, B=range" << std::endl;
  ret = test_solve<NumericT>(epsilon,
                             A, B_start, C_start,
                             arma_slice_A, arma_range_B, arma_range_C, arma_B
                            );
  if (ret != EXIT_SUCCESS)
    return ret;

  std::cout << "Now using A=slice, B=slice" << std::endl;
  ret = test_solve<NumericT>(epsilon,
                             A, B_start, C_start,
                             arma_slice_A, arma_slice_B, arma_slice_C, arma_B
                            );
  if (ret != EXIT_SUCCESS)
    return ret;

  return ret;

}



//
// Control functions
//

template< typename NumericT, typename Epsilon >
int test(Epsilon const& epsilon)
{
  int ret;

  std::cout << "////////////////////////////////" << std::endl;
  std::cout << "/// Now testing A=row, B=row ///" << std::endl;
  std::cout << "////////////////////////////////" << std::endl;
  ret = test_solve<NumericT, cuarma::row_major, cuarma::row_major>(epsilon);
  if (ret != EXIT_SUCCESS)
    return ret;


  std::cout << "////////////////////////////////" << std::endl;
  std::cout << "/// Now testing A=row, B=col ///" << std::endl;
  std::cout << "////////////////////////////////" << std::endl;
  ret = test_solve<NumericT, cuarma::row_major, cuarma::column_major>(epsilon);
  if (ret != EXIT_SUCCESS)
    return ret;

  std::cout << "////////////////////////////////" << std::endl;
  std::cout << "/// Now testing A=col, B=row ///" << std::endl;
  std::cout << "////////////////////////////////" << std::endl;
  ret = test_solve<NumericT, cuarma::column_major, cuarma::row_major>(epsilon);
  if (ret != EXIT_SUCCESS)
    return ret;

  std::cout << "////////////////////////////////" << std::endl;
  std::cout << "/// Now testing A=col, B=col ///" << std::endl;
  std::cout << "////////////////////////////////" << std::endl;
  ret = test_solve<NumericT, cuarma::column_major, cuarma::column_major>(epsilon);
  if (ret != EXIT_SUCCESS)
    return ret;



  return ret;
}

//
// -------------------------------------------------------------
//
int main()
{
   std::cout << std::endl;
   std::cout << "----------------------------------------------" << std::endl;
   std::cout << "----------------------------------------------" << std::endl;
   std::cout << "## Test :: BLAS 3 routines" << std::endl;
   std::cout << "----------------------------------------------" << std::endl;
   std::cout << "----------------------------------------------" << std::endl;
   std::cout << std::endl;

   int retval = EXIT_SUCCESS;

   std::cout << std::endl;
   std::cout << "----------------------------------------------" << std::endl;
   std::cout << std::endl;
   {
      typedef float NumericT;
      NumericT epsilon = NumericT(1.0E-3);
      std::cout << "# Testing setup:" << std::endl;
      std::cout << "  eps:     " << epsilon << std::endl;
      std::cout << "  numeric: float" << std::endl;
      retval = test<NumericT>(epsilon);
      if ( retval == EXIT_SUCCESS )
        std::cout << "# Test passed" << std::endl;
      else
        return retval;
   }
   std::cout << std::endl;
   std::cout << "----------------------------------------------" << std::endl;
   std::cout << std::endl;
   {
      {
        typedef double NumericT;
        NumericT epsilon = 1.0E-11;
        std::cout << "# Testing setup:" << std::endl;
        std::cout << "  eps:     " << epsilon << std::endl;
        std::cout << "  numeric: double" << std::endl;
        retval = test<NumericT>(epsilon);
        if ( retval == EXIT_SUCCESS )
          std::cout << "# Test passed" << std::endl;
        else
          return retval;
      }
      std::cout << std::endl;
      std::cout << "----------------------------------------------" << std::endl;
      std::cout << std::endl;
   }

   std::cout << std::endl;
   std::cout << "------- Test completed --------" << std::endl;
   std::cout << std::endl;


   return retval;
}
